
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void printf_from_gpu(void) {
	printf("Hello, world! From thread [%d, %d] From device\n", threadIdx.x, blockIdx.x);
}

int main(void) {
	printf("Hello, world from host!\n");
	printf_from_gpu<<<2,1>>>();
	hipDeviceSynchronize();
	return 0;
}
