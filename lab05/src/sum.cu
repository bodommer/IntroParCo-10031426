
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <unistd.h>

__global__ void VecAdd(int* A, int* B, int* C, int N) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N)
		C[i] = A[i] + B[i];
}

int main(int argc, char* argv[]) {
	int block = atoi(argv[2]);

	int power = atoi(argv[1]);
	int arr_size = pow(2, power);

	//printf("block count: %d, threads in block: %d, array_size: %d\n", block, arr_size/block, arr_size);

	srand(123);
	
	hipEvent_t start, stop;

	int* A, *B, *C, *D;
	A = (int*) malloc(sizeof(int) * arr_size);
	B = (int*) malloc(sizeof(int) * arr_size);
	C = (int*) malloc(sizeof(int) * arr_size);	
	D = (int*) malloc(sizeof(int) * arr_size);
	
	for (int i=0; i < arr_size; i++) {
		A[i] = rand() % (arr_size/2);
		B[i] = rand() % (arr_size/2);
		C[i] = 0;
		D[i] = A[i] + B[i];
	}

	int* devA, *devB, *devC;
	hipMalloc((void**)&devA, sizeof(int) * arr_size);
	hipMalloc((void**)&devB, sizeof(int) * arr_size);
	hipMalloc((void**)&devC, sizeof(int) * arr_size);

	hipMemcpy(devA, A, sizeof(int) * arr_size, hipMemcpyHostToDevice);
        hipMemcpy(devB, B, sizeof(int) * arr_size, hipMemcpyHostToDevice);	
	
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	VecAdd<<<block,arr_size/block>>>(devA, devB, devC, arr_size);
	hipError_t kernel_error = hipGetLastError();
	hipEventRecord(stop);

	float runtime = 0.0f;
	hipEventSynchronize(stop);
	hipEventElapsedTime(&runtime, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	if (kernel_error != hipSuccess) {
		printf("LastError: %s\nDetails: %s\n", hipGetErrorName(kernel_error), hipGetErrorString(kernel_error));
		printf("ERROR: array %d - %d blocks\n", arr_size, block);
	} else {
		hipMemcpy(C, devC, sizeof(int) *arr_size, hipMemcpyDeviceToHost);

		for (int i=0; i < 5; i++) {
			int r = rand() % arr_size;
			//printf("r: %d, C: %d, D: %d\n", r, C[r], D[r]);
			assert(C[r] == D[r]);
		}
		printf("%d;%d;%f\n", arr_size, block, runtime);
		//printf("OK: array %d - %d blocks\n", arr_size, block);
	}

	free(A);
	free(B);
	free(C);
	free(D);
	hipFree(devA);
	hipFree(devB);
	hipFree(devC);

	return kernel_error;
}
